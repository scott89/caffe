#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {


template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + bottom[i]->offset(n), weight,
          top_data + top[i]->offset(n));
      if (this->has_mask_ && this->phase_ == TRAIN) {
	const unsigned int* mask = this->mask_.gpu_data();
	this->forward_gpu_mask(top_data + top[i]->offset(n), mask);
      }
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + top[i]->offset(n), bias);
      }
    }
  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  //LOG(INFO) << "start of convolutionlayer backward_gpu";
  //CHECK((this->kstride_h_ == 1) && (this->kstride_w_ == 1)) << "Backward_gpu is not implemented for fully convolutin.";
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    Dtype* top_diff_mutable = top[i]->mutable_gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + top[i]->offset(n));
      }
    }
    // Mask
    if (this->has_mask_ && this->phase_ == TRAIN) {
      const unsigned int* mask = this->mask_.gpu_data();
      for (int n = 0; n < this->num_; ++n) {
	this->backward_gpu_mask(top_diff_mutable + top[i]->offset(n), mask);
      }
    }
      
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
	  if (this->kstride_h_ == 1) {
	    this->weight_gpu_gemm(bottom_data + bottom[i]->offset(n),
              top_diff + top[i]->offset(n), weight_diff);
	  } else {
	    this->fcn_weight_gpu_gemm(bottom_data + bottom[i]->offset(n),
              top_diff + top[i]->offset(n), weight_diff);
	    //LOG(INFO) << "fcn_weight_gpu_gemm";
	  }
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + top[i]->offset(n), weight,
              bottom_diff + bottom[i]->offset(n));
        }
      }
    }
  }
  //LOG(INFO) << "end of convolutionlayer backward_gpu";
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
